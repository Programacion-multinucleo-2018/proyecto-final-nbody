#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include "main.h"

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////

__global__ void calculate_acceleration(Vertex *v, unsigned int n) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < n && j < n && i != j) {
    float distance = sqrt(pow(v[i].position.x - v[j].position.x, 2) +
                          pow(v[i].position.y - v[j].position.y, 2) +
                          pow(v[i].position.z - v[j].position.z, 2));

    float magnitude = G_CONSTANT / pow(distance, 3);

    float3 vector;
    vector.x = magnitude * (v[i].position.x - v[j].position.x);
    vector.y = magnitude * (v[i].position.y - v[j].position.y);
    vector.z = magnitude * (v[i].position.z - v[j].position.z);

    atomicAdd(&(v[i].acceleration.x), -vector.x * v[j].mass);
    atomicAdd(&(v[i].acceleration.y), -vector.y * v[j].mass);
    atomicAdd(&(v[i].acceleration.z), -vector.z * v[j].mass);

    atomicAdd(&(v[j].acceleration.x), vector.x * v[i].mass);
    atomicAdd(&(v[j].acceleration.y), vector.y * v[i].mass);
    atomicAdd(&(v[j].acceleration.z), vector.z * v[i].mass);
  }
}

__global__ void calculate_position(Vertex *v, unsigned int n, float delta) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    v[i].speed.x += v[i].acceleration.x * delta;
    v[i].speed.y += v[i].acceleration.y * delta;
    v[i].speed.z += v[i].acceleration.z * delta;

    v[i].position.x += v[i].speed.x * delta;
    v[i].position.y += v[i].speed.y * delta;
    v[i].position.z += v[i].speed.z * delta;

    v[i].acceleration.x = 0.0f;
    v[i].acceleration.y = 0.0f;
    v[i].acceleration.z = 0.0f;

    printf("#%i\nSpeed: %f %f %f\nPosition: %f %f %f\n", i, v[i].speed.x,
           v[i].speed.y, v[i].speed.z, v[i].position.x, v[i].position.y,
           v[i].position.z);
  }
}

int main(int argc, const char **argv) {
  int n_vertices = 2;
  float delta = 1.0f;
  Vertex *v = new Vertex[2];

  v[0].mass = 5.972e+24;

  v[0].position.x = 0.0f;
  v[0].position.y = 0.0f;
  v[0].position.z = 0.0f;
  v[0].position.w = 1.0f;

  v[0].speed.x = 0.0f;
  v[0].speed.y = 0.0f;
  v[0].speed.z = 0.0f;

  v[0].acceleration.x = 0.0f;
  v[0].acceleration.y = 0.0f;
  v[0].acceleration.z = 0.0f;

  v[1].mass = 1;

  v[1].position.x = 6000000;
  v[1].position.y = 0.0f;
  v[1].position.z = 0.0f;
  v[1].position.w = 1.0f;

  v[1].speed.x = 0.0f;
  v[1].speed.y = 0.0f;
  v[1].speed.z = 0.0f;

  v[1].acceleration.x = 0.0f;
  v[1].acceleration.y = 0.0f;
  v[1].acceleration.z = 0.0f;

  Vertex *d_v;
  hipMalloc(&d_v, sizeof(Vertex) * 2);
  hipMemcpy(d_v, v, sizeof(Vertex) * 2, hipMemcpyHostToDevice);

  // launchKernel (devPtr, DIM, dt);
  dim3 numBlocks((int)ceil((float)n_vertices / 16.0),
                 (int)ceil((float)n_vertices / 16.0));
  dim3 numThreads(16, 16);
  calculate_acceleration<<<numBlocks, numThreads>>>(d_v, n_vertices);
  numBlocks.y = 1;
  numThreads.y = 1;
  calculate_position<<<numBlocks, numThreads>>>(d_v, n_vertices, delta);

  // unmapping our shared resource. This call is important to make prior to
  // performing rendering tasks because it provides synchronization between the
  // CUDA and graphics portions of the application. Specifically, it implies
  // that all CUDA operations performed prior to the call to
  // hipGraphicsUnmapResources() will complete before ensuing graphics calls
  // begin.
  hipFree(d_v);
}

void unregRes(hipGraphicsResource **res) {
  checkCudaErrors(hipGraphicsUnmapResources(1, res, NULL));
}

void chooseDev(int ARGC, const char **ARGV) { gpuGLDeviceInit(ARGC, ARGV); }

void regBuffer(hipGraphicsResource **res, unsigned int &vbo) {
  // setting up graphics interoperability by notifying the CUDA runtime
  // that we intend to share the OpenGL buffer named vbo with CUDA.
  checkCudaErrors(
      hipGraphicsGLRegisterBuffer(res, vbo, cudaGraphicsMapFlagsWriteDiscard));
}